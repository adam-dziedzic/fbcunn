#include <hipDNN.h>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <opencv2/opencv.hpp>

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }                                                          \


static void HandleError(hipError_t err,
                        const char *file,
                        int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))


cv::Mat load_image(const char *image_path) {
    cv::Mat image = cv::imread(image_path, CV_LOAD_IMAGE_COLOR);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    std::cerr << "Input Image: " << image.rows << " x " << image.cols << " x "
              << image.channels() << std::endl;
    return image;
}

void save_image(const char *output_filename,
                float *buffer,
                int height,
                int width) {
    cv::Mat output_image(height, width, CV_32FC3, buffer);
    // Make negative values zero.
    cv::threshold(output_image,
                  output_image,
            /*threshold=*/0,
            /*maxval=*/0,
                  cv::THRESH_TOZERO);
    cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
    output_image.convertTo(output_image, CV_8UC3);
    cv::imwrite(output_filename, output_image);
    std::cerr << "Wrote output to " << output_filename << std::endl;
}

int main(int argc, const char *argv[]) {
    if (argc < 2) {
        std::cerr << "usage: conv <image> [gpu=0] [sigmoid=0]" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    int gpu_id = (argc > 2) ? std::atoi(argv[2]) : 0;
    std::cerr << "GPU: " << gpu_id << std::endl;

    bool with_sigmoid = (argc > 3) ? std::atoi(argv[3]) : 0;
    std::cerr << "With sigmoid: " << std::boolalpha << with_sigmoid
              << std::endl;


    int in_batch_size = 1;
    int out_channels = 3;
    int kernel_height = 3;
    int kernel_width = 3;
    int pad_height = 1;
    int pad_width = 1;
    int vertical_stride = 1;
    int horizontal_stride = 1;

    bool tests = false;
    if (tests == true) {
        in_batch_size = 32;
        out_channels = 64;
        kernel_height = 7;
        kernel_width = 7;
        pad_height = 3;
        pad_width = 3;
        vertical_stride = 2;
        horizontal_stride = 2;
    }

    cv::Mat image = load_image(argv[1]);

    hipSetDevice(gpu_id);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/in_batch_size,
            /*channels=*/3,
            /*image_height=*/image.rows,
            /*image_width=*/image.cols));

    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/out_channels,
            /*in_channels=*/3,
            /*kernel_height=*/kernel_height,
            /*kernel_width=*/kernel_width));

    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
            /*pad_height=*/pad_height,
            /*pad_width=*/pad_width,
            /*vertical_stride=*/vertical_stride,
            /*horizontal_stride=*/horizontal_stride,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
            /*computeType=*/HIPDNN_DATA_FLOAT));

    int batch_size{0}, channels{0}, height{0}, width{0};
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     &batch_size,
                                                     &channels,
                                                     &height,
                                                     &width));

    std::cerr << "Output Image: " << height << " x " << width << " x "
              << channels
              << std::endl;

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
            /*format=*/HIPDNN_TENSOR_NHWC,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/3,
            /*image_height=*/image.rows,
            /*image_width=*/image.cols));

    const int requestedAlgoCount = 8;
    int returnedAlgoCount = 8;
//  hipdnnConvolutionFwdAlgoPerf_t perfResults;
    hipdnnConvolutionFwdAlgoPerf_t perfResults[requestedAlgoCount];
//  HANDLE_ERROR(hipMalloc((void**)&perfResults,
//          sizeof(hipdnnConvolutionFwdAlgoPerf_t) * requestedAlgoCount));

    checkCUDNN(hipdnnFindConvolutionForwardAlgorithm(
            /*hipdnnHandle_t*/cudnn,
            /*hipdnnTensorDescriptor_t*/input_descriptor,
            /*hipdnnFilterDescriptor_t*/kernel_descriptor,
            /*hipdnnConvolutionDescriptor_t*/convolution_descriptor,
            /*hipdnnTensorDescriptor_t*/output_descriptor,
            /*int*/requestedAlgoCount,
            /*int**/&returnedAlgoCount,
            /*hipdnnConvolutionFwdAlgoPerf_t*/perfResults));

    std::cout << "returnedAlgoCount: " << returnedAlgoCount << std::endl;

    for (int i = 0; i < returnedAlgoCount; ++i) {
        std::cout << "perfResults: " << i << " algo " << perfResults[i].algo
                  << " time: " << perfResults[i].time
                  << " memory: " << perfResults[i].memory
                  << " status: " << perfResults[i].status
                  << " determinism: " << perfResults[i].determinism
                  << " math_type: " << perfResults[i].mathType
                  << std::endl;
    }

//  for (int i = 0; i < returnedAlgoCount; ++i) {
//      std::cout << "# convolution algorithm: " << d_perfResults[i].algo
//                << " time: " << d_perfResults[i].time
//                << " memory: " << d_perfResults[i].memory
//                << std::endl;
//  }

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(
            hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                                input_descriptor,
                                                kernel_descriptor,
                                                convolution_descriptor,
                                                output_descriptor,
                                                HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                    /*memoryLimitInBytes=*/0,
                                                &convolution_algorithm));

    // https://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnConvolutionFwdAlgo_t
    std::cout << "# convolution algorithm: " << convolution_algorithm
              << std::endl;

    size_t workspace_bytes{0};
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       input_descriptor,
                                                       kernel_descriptor,
                                                       convolution_descriptor,
                                                       output_descriptor,
                                                       convolution_algorithm,
                                                       &workspace_bytes));
    std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB"
              << std::endl;
    assert(workspace_bytes > 0);

    void *d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);

    int image_bytes = batch_size * channels * height * width * sizeof(float);

    float *d_input{nullptr};
    hipMalloc(&d_input, image_bytes);
    hipMemcpy(d_input, image.ptr<float>(0), image_bytes,
               hipMemcpyHostToDevice);

    float *d_output{nullptr};
    hipMalloc(&d_output, image_bytes);
    hipMemset(d_output, 0, image_bytes);

    // clang-format off
    const float kernel_template[3][3] = {
            {1, 1,  1},
            {1, -8, 1},
            {1, 1,  1}
    };
    // clang-format on

    float h_kernel[3][3][3][3];
    for (int kernel = 0; kernel < 3; ++kernel) {
        for (int channel = 0; channel < 3; ++channel) {
            for (int row = 0; row < 3; ++row) {
                for (int column = 0; column < 3; ++column) {
                    h_kernel[kernel][channel][row][column] = kernel_template[row][column];
                }
            }
        }
    }

    float *d_kernel{nullptr};
    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

    const float alpha = 1.0f, beta = 0.0f;

    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       input_descriptor,
                                       d_input,
                                       kernel_descriptor,
                                       d_kernel,
                                       convolution_descriptor,
                                       convolution_algorithm,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       output_descriptor,
                                       d_output));

    if (with_sigmoid) {
        hipdnnActivationDescriptor_t activation_descriptor;
        checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
        checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
                                                HIPDNN_ACTIVATION_SIGMOID,
                                                HIPDNN_PROPAGATE_NAN,
                /*relu_coef=*/0));
        checkCUDNN(hipdnnActivationForward(cudnn,
                                          activation_descriptor,
                                          &alpha,
                                          output_descriptor,
                                          d_output,
                                          &beta,
                                          output_descriptor,
                                          d_output));
        hipdnnDestroyActivationDescriptor(activation_descriptor);
    }

    float *h_output = new float[image_bytes];
    hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);

    save_image("cudnn-out.png", h_output, height, width);

    delete[] h_output;
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
//  hipFree(d_perfResults);
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);
}